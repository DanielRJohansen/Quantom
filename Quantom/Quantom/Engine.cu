#include "hip/hip_runtime.h"
#include "Engine.cuh"




Simulation* Engine::prepSimulation(Simulation* simulation) {
	this->simulation = simulation;
	srand(290128301);
	boxbuilder.build(simulation);
	printf("Boxbuild complete!\n");

	updateNeighborLists();
	printf("Neighborlists ready\n");



	simulation->moveToDevice();
	return this->simulation;
}





void Engine::updateNeighborLists() {	// Write actual function later;
	int maxc = 1'000'000; // this is temporary!
	CompoundState* statebuffer_host = new CompoundState[maxc];
	CompoundNeighborInfo* neighborlists_host = new CompoundNeighborInfo[maxc];
	hipMemcpy(statebuffer_host, simulation->box->compound_state_buffer, sizeof(CompoundState) * maxc, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	// This only needs to be done the first time... Or does it????
	for (int i = 0; i < maxc; i++) {
		neighborlists_host[i].n_neighbors = 0;
	}
		
	printf("1\n");

	// This is the temp func //
	for (int i = 0; i < simulation->box->n_compounds; i++) {
		for (int j = 0; j < simulation->box->n_compounds; j++) {
			if (i != j) {
				CompoundNeighborInfo* nlist = neighborlists_host;
				nlist->neighborcompound_indexes[nlist->n_neighbors++] = j;
			}
		}
	}
	// --------------------- //

	hipMemcpy(simulation->box->compound_neighborinfo_buffer, neighborlists_host, sizeof(CompoundNeighborInfo) * maxc, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
}






//--------------------------------------------------------------------------	SIMULATION BEGINS HERE --------------------------------------------------------------//


void Engine::step() {
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Error before step!");
		exit(1);
	}


	auto t0 = std::chrono::high_resolution_clock::now();


	int compounds_per_sm = 1000;
	Box* box = simulation->box;	// Why the fuck do i have to do this, VisualStudio???!
	for (int i = 0; i < N_STREAMS; i++) {
		int offset = i * compounds_per_sm;
		//intramolforceKernel <<< compounds_per_sm, 3, 0, stream[i] >>> (box, offset);
	}
	forceKernel <<< box->n_compounds, 256 >>> (box);
	hipDeviceSynchronize();


	auto t1 = std::chrono::high_resolution_clock::now();



	int blocks_handled = 0;
	while (blocks_handled < sim_blocks) {
		for (int i = 0; i < N_STREAMS; i++) {
			//stepKernel << < BLOCKS_PER_SM, MAX_FOCUS_BODIES, 0, stream[i] >> > (simulation, blocks_handled);
			blocks_handled += BLOCKS_PER_SM;
			if (blocks_handled >= sim_blocks)
				break;
		}

		hipDeviceSynchronize();
		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "Error during step :/\n");
			exit(1);
		}
	}




	auto t2 = std::chrono::high_resolution_clock::now();

	blocks_handled = 0;
	while (blocks_handled < sim_blocks) {
		for (int i = 0; i < N_STREAMS; i++) {
			//updateKernel << < BLOCKS_PER_SM, dim3(3,3,3), 0, stream[i] >> > (simulation, blocks_handled);
			blocks_handled += BLOCKS_PER_SM;
			if (blocks_handled >= sim_blocks)
				break;
		}

		hipDeviceSynchronize();
		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "Error during update :/\n");
			exit(1);
		}
	}

	auto t3 = std::chrono::high_resolution_clock::now();

	bool verbose = true;
	if (verbose) {
		int intra_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
		int inter_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
		int update_duration = (int)std::chrono::duration_cast<std::chrono::microseconds>(t3 - t2).count();
		timings = timings + Int3(intra_duration, inter_duration, update_duration);
		//printf("\nStep %d ys.\tUpdate: %d\n\n", step_duration, update_duration);
	}


	simulation->step++;
}





// ------------------------------------------------------------------------------------------- DEVICE FUNCTIONS -------------------------------------------------------------------------------------------//




__device__ float cudaMax(float a, float b) {
	if (a > b)
		return a;
	return b;
}
__device__ float cudaMin(float a, float b) {
	if (a < b)
		return a;
	return b;
}



																				// TODO: MAKE IS SUCH THAT A BODY CAN NEVER BE EXACTLY ON THE EDGE OF FOCUS, THUS APPEARING IN MULTIPLE GROUPS!



__device__ Float3 getHyperPosition(Float3 pos) {			// Dont thj�nk this is right anymore??????
	pos = pos + Float3(BOX_LEN, BOX_LEN, BOX_LEN) * 1.5;
	pos = pos.elementwiseModulus(BOX_LEN);
	return pos - Float3(BOX_LEN, BOX_LEN, BOX_LEN) * 0.5;
}

__device__ Float3 getClosestMirrorPos(Float3 pos, Float3 block_center) {	// Block center can also just be whatever particle you want the pos to be close to...
	Float3 dists = block_center - pos;
	Float3 abs_dists = dists.abs();
	abs_dists = abs_dists + Float3(0.000001 * !(abs_dists.x), 0.000001 * !(abs_dists.y), 0.000001 * !(abs_dists.z));
	Float3 directional_hotencoded_vector(
		round(dists.x / abs_dists.x) * (abs_dists.x > BOX_LEN_HALF),
		round(dists.y / abs_dists.y) * (abs_dists.y > BOX_LEN_HALF),
		round(dists.z / abs_dists.z) * (abs_dists.z > BOX_LEN_HALF)
	);
	return pos + directional_hotencoded_vector * Float3(BOX_LEN, BOX_LEN, BOX_LEN);
}


__device__ float getAngle(Float3 v1, Float3 v2) {
	return acos((v1.dot(v2)) / (v1.len() * v2.len()));
}


/*
constexpr float sigma = 0.3923;	//nm
constexpr float epsilon = 0.5986 * 1'000; //kJ/mol | J/mol
__device__ Float3 calcLJForce(Particle* particle0, Particle* particle1, int i, int bid) {	// Applying force to p0 only! 
	float dist = (particle0->pos - particle1->pos).len();
	float fraction = sigma / dist;

	float f2 = fraction * fraction;
	float f6 = f2 * f2 * f2;
	float f12 = f6 * f6;

	float LJ_pot = 4 * epsilon * (f12 - f6);
	Float3 force_unit_vector = (particle0->pos - particle1->pos).norm();	// + is repulsive, - is attractive


	if (LJ_pot > 20'000) {
		//body0->molecule_type = 99;
		printf("\n\n KILOFORCE! Block %d thread %d\n", bid, threadIdx.x);
		printf("other body index: %d\n", i);
		printf("Body 0 id: %d     %f %f %f\n", particle0->id, particle0->pos.x, particle0->pos.y, particle0->pos.z);
		printf("Body 1 id: %d     %f %f %f\n", particle1->id, particle1->pos.x, particle1->pos.y, particle1->pos.z);
		printf("Distance: %f\n", (particle0->pos - particle1->pos).len());

	}
	return force_unit_vector * LJ_pot;
}
*/


/*
constexpr float kb = 17.5 * 10e+6;		//	J/(mol*nm^2)
__device__ void calcPairbondForce(Compound_H2O* compound, PairBond* pairbond, float* dataptr) {
	Float3 particle1_mirrorpos = getClosestMirrorPos(compound->particles[pairbond->atom_indexes[1]].pos, compound->particles[pairbond->atom_indexes[0]].pos);
	Float3 direction = compound->particles[pairbond->atom_indexes[0]].pos - particle1_mirrorpos;
	//int focus_index = pairbond->atom_indexes[1] == threadIdx.x;	// If it is not pos 1 we get false, meaning pos 0... Not beautiful but it works.
	//direction = direction - direction * 2 * (1 * focus_index);	// Flip dir if focusatom is at index 1

	if (pairbond->atom_indexes[1] == threadIdx.x)	// Flip so we repel the particle in focus
		direction = direction * -1;
	

	float dist = direction.len();
	float dif = dist - pairbond->reference_dist;
	if (dif > 0)	// Flip to attraction
		direction = direction * -1;

	float force_scalar = 0.5 * kb * (dif * dif);
	direction = direction.norm();
	//float invert_if_attraction = -1 + (2 * (dif < 0));

	compound->particles[threadIdx.x].force = compound->particles[threadIdx.x].force + direction * force_scalar;

	if (compound->startindex_particle + threadIdx.x == LOG_P_ID) {
		*dataptr = dist;
	}
		
	Float3 p0p = compound->particles[pairbond->atom_indexes[0]].pos;
	if (force_scalar > WARN_FORCE) {
		printf("\n\n Atom id %d dist %f dif: %f FORCE %f Repulsive %d\n", compound->startindex_particle + threadIdx.x, dist, dif, force_scalar, dif < 0);
		(direction * force_scalar).print('b');
		compound->particles[threadIdx.x].force.print('B');
		//printf("p0 %f %f %f \tp1_mirror %f %f %f\n", p0p.x, p0p.y, p0p.z, particle1_mirrorpos.x, particle1_mirrorpos.y, particle1_mirrorpos.z);
	}
}


constexpr float ktheta = 65 * 10e+3;	// J/mol
__device__ void calcAngleForce(Compound_H2O* compound, AngleBond* anglebond, float* dataptr) {	// We fix the middle particle and move the other particles so they are closest as possible
	Float3 p0_mirrorpos = getClosestMirrorPos(compound->particles[anglebond->atom_indexes[0]].pos, compound->particles[anglebond->atom_indexes[1]].pos);
	Float3 p2_mirrorpos = getClosestMirrorPos(compound->particles[anglebond->atom_indexes[2]].pos, compound->particles[anglebond->atom_indexes[1]].pos);

	Float3 v1 = p0_mirrorpos - compound->particles[anglebond->atom_indexes[1]].pos;
	Float3 v2 = p2_mirrorpos - compound->particles[anglebond->atom_indexes[1]].pos;

	Float3 force_direction = p0_mirrorpos-p2_mirrorpos;
	force_direction = force_direction - force_direction * 2 * (threadIdx.x == 2);
	force_direction = force_direction.norm();

	float angle = getAngle(v1, v2);
	//printf("\nangle %f\n", angle);
	float dif = angle - anglebond->reference_theta;
	float force_scalar = 0.5 * ktheta * (dif * dif);




	float invert_if_attraction = -1 + (2 * (dif < 0));

	compound->particles[threadIdx.x].force = compound->particles[threadIdx.x].force + force_direction * force_scalar * invert_if_attraction;


	if (force_scalar > WARN_FORCE || abs(angle) < 0.1) {
		printf("\n####################\n####################\n####################");
		printf("\nParticle ID %d Angle %f Force %f\n", compound->startindex_particle + threadIdx.x, angle, force_scalar);
		(force_direction * force_scalar * invert_if_attraction).print('a');
	}
	

	if (compound->startindex_particle + threadIdx.x == LOG_P_ID) {
		*dataptr = angle;
	}		
}
*/
__device__ void integrateTimestep(CompactParticle* particle, Float3 force, float dt) {	// Kinetic formula: v = sqrt(2*K/m), m in kg
	Float3 vel_next = particle->vel_prev + (force * (1000.f/particle->mass) * dt);
	particle->pos = particle->pos + vel_next * dt;
	particle->vel_prev = vel_next;
}

// ------------------------------------------------------------------------------------------- KERNELS -------------------------------------------------------------------------------------------//

__global__ void forceKernel(Box* box) {
	__shared__ Compound_H2O compound;
	//__shared__ Compound_H2O_

	if (threadIdx.x == 0) {
		compound = box->compounds[blockIdx.x];
	}
	__syncthreads();
	bool thread_compound_active = (compound.n_particles > threadIdx.x);




	if (thread_compound_active) {
		if (threadIdx.x == 0) {
			//compound.particles[threadIdx.x].pos.print();
			box->compound_state_buffer[blockIdx.x].particle_cnt = compound.n_particles;
		}
			
		box->compound_state_buffer[blockIdx.x].positions[threadIdx.x] = compound.particles[threadIdx.x].pos;
	}
		
	//box->compounds[blockIdx.x].particles[threadIdx.x].pos = compound.particles[threadIdx.x].pos;
}



































/*
__global__ void intramolforceKernel(Box* box, int offset) {	// 1 thread per particle in compound
	__shared__ Compound_H2O compound;

	uint32_t compound_index = blockIdx.x + offset;
	if (compound_index >= box->n_compounds)
		return;



	if (threadIdx.x == 0) {
		compound = box->compounds[compound_index];
	}
	__syncthreads();
	compound.particles[threadIdx.x].pos = box->particles[compound.startindex_particle + threadIdx.x].pos;
	compound.particles[threadIdx.x].force = Float3(0, 0, 0);
	__syncthreads();

	for (int i = 0; i < compound.n_pairbonds; i++) {	// Bond forces
		PairBond* bond = &compound.pairbonds[i];
		if (bond->atom_indexes[0] == threadIdx.x || bond->atom_indexes[1] == threadIdx.x) {
			calcPairbondForce(&compound, bond, &box->outdata1[box->data1_cnt]);
			if (compound.startindex_particle + threadIdx.x == LOG_P_ID)
				box->data1_cnt++;
		}
	}

	for (int i = 0; i < compound.n_anglebonds; i++) {	// Angle forces
		AngleBond* bond = &compound.anglebonds[i];
		if (bond->atom_indexes[0] == threadIdx.x || bond->atom_indexes[2] == threadIdx.x) {
			//calcAngleForce(&compound, bond, &box->outdata2[box->data2_cnt]);
			if (compound.startindex_particle + threadIdx.x == LOG_P_ID)
				box->data2_cnt++;
		}
	}
	//CompactParticle* particle = &compound.particles[threadIdx.x];
	//PairBond* pairbond = &compound.pairbonds[threadIdx.x];
	//calcPairbondForce(&compound, pairbond);	// This applies the force directly to the particles

	box->particles[compound.startindex_particle + threadIdx.x].force = compound.particles[threadIdx.x].force;
}






__global__ void stepKernel(Simulation* simulation, int offset) {
	int blockID = blockIdx.x + offset;	// Maybe save the register, and just calculate it a couple times.
	int bodyID = threadIdx.x;

	if (blockID >= simulation->box->n_blocks)
		return;
	


	// Load bodies into shared memory
	__shared__ Block block;	
	__shared__ AccessPoint accesspoint;
	
	if (threadIdx.x == 0) {
		block = simulation->box->blocks[blockID];
		accesspoint = AccessPoint();
	}
		
	__syncthreads();
	Particle particle = block.focus_particles[bodyID];

	

	// --------------------------------- ACTUAL MOLECULAR DYNAMICS HAPPENING HERE! --------------------------------- //
	// Calc all Lennard-Jones forces from focus bodies
	if (particle.active) {						// This part acounts for about 2/5 of compute time
		// I assume that all present molecules come in order!!!!!!



		particle.force = simulation->box->particles[particle.id].force;
		simulation->box->blocks[blockID].focus_particles[bodyID].color[2] = simulation->box->particles[particle.id].color[2];
		Float3 force_total;
		for (int i = 0; i < MAX_FOCUS_BODIES; i++) {
			if (block.focus_particles[i].active) {
				if (i != bodyID && particle.compoundID !=  block.focus_particles[i].compoundID) {
					force_total = force_total + calcLJForce(&particle, &block.focus_particles[i], -i, blockID);
				}
			}
			else {
				break;
			}
		}

		// Calc all forces from Near bodies
		for (int i = 0; i < MAX_NEAR_BODIES; i++) {
			if (block.near_particles[i].active && particle.compoundID != block.near_particles[i].compoundID) {
				force_total = force_total + calcLJForce(&particle, &block.near_particles[i], i + MAX_FOCUS_BODIES, blockID);
			}
			else {
				break;
			}
		}
		


		//particle.force = simulation->box->particles[particle.id].force;
		//printf("\n ID %d\tInter: %f %f %f\tIntra %f %f %f\n", particle.id, force_total.x, force_total.y, force_total.z, particle.force.x, particle.force.y, particle.force.z);
		

		//force_total = force_total + particle.force;
		particle.force = particle.force + force_total;
		
		if (particle.id == LOG_P_ID) {
			//simulation->box->outdata3[simulation->box->data3_cnt++] = force_total.len();
			simulation->box->outdata3[simulation->box->data3_cnt++] = force_total.len();
			simulation->box->outdata4[simulation->box->data4_cnt++] = particle.force.len();
		}

		if (particle.force.len() > WARN_FORCE) {
			printf("\n");
			particle.force.print('I');
			force_total.print('T');
		}
		if (particle.force.len() > END_SIM_FORCE) {
			simulation->finished = true;
			printf("Ending due to particle %d\n", particle.id);
		}


		// Integrate position  AFTER ALL BODIES IN BLOCK HAVE BEEN CALCULATED? No should not be a problem as all update their local body, 
		// before moving to shared?? Although make the local just a pointer might be faster, since a SimBody might not fit in thread registers!!!!!
		integrateTimestep(simulation, &particle);




		// Correct for bonded-body constraints? Or maybe this should be calculated as a force too??




		
		// Swap with mirror image if body has moved out of box
		Float3 hyper_pos = getHyperPosition(particle.pos);
		if ((hyper_pos - particle.pos).len() > 1) {	// If the hyperposition is different, the body is out, and we import the mirror body
			//printf("\nSwapping Body %f %f %f to hyperpos %f %f %f\n\n", body.pos.x, body.pos.y, body.pos.z, hyper_pos.x, hyper_pos.y, hyper_pos.z);
			particle.pos = hyper_pos;
		}	
		simulation->box->particles[particle.id].pos = particle.pos;

	}






	// Publish new positions for the focus bodies
	accesspoint.particles[bodyID] = particle;

	// Mark all bodies as obsolete
	simulation->box->blocks[blockID].focus_particles[bodyID].active = false;	// Need to run update kernel before rendering, or no particle will be rendered.


	__syncthreads();
	if (bodyID == 0) {
		simulation->box->accesspoint[blockID] = accesspoint;
	}
} 












__global__ void updateKernel(Simulation* simulation, int offset) {
	int blockID1 = blockIdx.x + offset;
	if (blockID1 >= simulation->box->n_blocks)
		return;

	int threadID1 = indexConversion(Int3(threadIdx.x, threadIdx.y, threadIdx.z), 3);

	

	
	__shared__ Float3 block_center;
	__shared__ Int3 blockID3;
	__shared__ int bpd;
	__shared__ char element_cnt_focus[27];
	__shared__ char element_sum_focus[27 + 1];
	__shared__ short int element_cnt_near[27];
	__shared__ short int element_sum_near[27+1];
	__shared__ char relation_array[27][MAX_FOCUS_BODIES];	// 0 = far, 1 = near, 2 = focus
	

	

	//element_sum_focus[threadID1 + 1] = -1;
	//element_sum_near[threadID1 + 1] = -1;
	for (int i = 0; i < MAX_FOCUS_BODIES; i++)
		relation_array[threadID1][i] = 0;
	if (threadID1 == 0) {
		block_center = simulation->box->blocks[blockID1].center;			// probably faster to just calculate
		bpd = simulation->blocks_per_dim;
		blockID3 = indexConversion(blockID1, bpd);
		element_sum_focus[0] = 0;
		element_sum_near[0] = 0;	
	}
	
	__syncthreads();


	AccessPoint accesspoint;		// Personal to all threads
		
		Int3 neighbor_index3 = blockID3 + (Int3(threadIdx.x, threadIdx.y, threadIdx.z) - Int3(1, 1, 1));
		neighbor_index3 = neighbor_index3 + Int3(bpd * (neighbor_index3.x == -1), bpd * (neighbor_index3.y == -1), bpd * (neighbor_index3.z == -1));
		neighbor_index3 = neighbor_index3 - Int3(bpd * (neighbor_index3.x == bpd), bpd * (neighbor_index3.y == bpd), bpd * (neighbor_index3.z == bpd));
		accesspoint = simulation->box->accesspoint[indexConversion(neighbor_index3, bpd)];
	
	

	{	// To make these to variables temporary
		int focus_cnt = 0;
		int near_cnt = 0;
		int array_index = threadID1 + 1;
		for (int i = 0; i < MAX_FOCUS_BODIES; i++) {
			Particle* particle = &accesspoint.particles[i];
			if (!particle->active)
				break;

			Float3 closest_mirror_pos = getClosestMirrorPos(particle->pos, block_center);
			int relation_type = (bodyInNear(&closest_mirror_pos, &block_center) + bodyInFocus(&particle->pos, &block_center) * 2);
			



			if (relation_type == 1)		// If the body is ONLY in near, we make a temporary copy, with the mirrors position.
				particle->pos = closest_mirror_pos;

			//if (relation_type > 1 && body->id == 0)
				//printf("\nLoading %d to block %d %d %d by thread %d %d %d from block %d %d %d\n", body->id, blockID3.x, blockID3.y, blockID3.z, threadIdx.x-1, threadIdx.y - 1, threadIdx.z - 1, neighbor_index3.x, neighbor_index3.y, neighbor_index3.z);

			relation_array[threadID1][i] = relation_type;
			near_cnt += (relation_type == 1);
			focus_cnt += (relation_type > 1);
		}

		for (int i = 0; i < 27; i++) {	// Reserve spaces 
			if (threadID1 == i) {
				element_sum_focus[array_index] = element_sum_focus[array_index-1] + focus_cnt;
				element_sum_near[array_index] = element_sum_near[array_index-1] + near_cnt;
				if (element_sum_focus[array_index] >= MAX_FOCUS_BODIES || element_sum_near[array_index] >= MAX_NEAR_BODIES) {
					printf("Block %d overflowing! near %d    focus %d\n", blockID1, element_sum_near[i + 1], element_sum_focus[i + 1]);
					simulation->finished = true;
					break;
				}
			}
			__syncthreads();
		}
		
	}

	
	{
		int focus_index = element_sum_focus[threadID1];
		int near_index = element_sum_near[threadID1];
		for (int i = 0; i < MAX_FOCUS_BODIES; i++) {
			if (relation_array[threadID1][i] > 1) 
				simulation->box->blocks[blockID1].focus_particles[focus_index++] = accesspoint.particles[i];
			else if (relation_array[threadID1][i] == 1) {
				simulation->box->blocks[blockID1].near_particles[near_index++] = accesspoint.particles[i];
				//printf("\n%d\t %d %d %d\t%d\t nearindex: %d\n", blockID1, blockID3.x, blockID3.y, blockID3.z, accesspoint.bodies[i].molecule_type, near_index);
			}	
		}



		// Handle deactivating now-obsolete bodies
		// The stepkernel will handle the marking of focus bodies, as it has the correct amount of threads. Less overhead!

		// For nearbodies we only need to terminate 1 body, this saves alot of writes to global!
		if (threadID1 == 26) {	
			if (near_index < (MAX_NEAR_BODIES))
				simulation->box->blocks[blockID1].near_particles[near_index].active = false;
			//printf("\n%f %f %f\tExporting %d\n", block_center.x, block_center.y, block_center.z, near_index);
			//printf("Block %d loaded %d bodies\n", blockID1, focus_index);
		}
	}
}

*/

