#include "hip/hip_runtime.h"
#include "Raytracer.cuh"



Ray::Ray(Float3 unit_vector, Float3 origin) : unit_vector(unit_vector), origin(origin) {}

__device__ void Ray::findBlockHits(Box* box, Float3 focalpoint) {
    for (int i = 0; i < MAX_RAY_BLOCKS; i++)
		block_indexes[i] = -1;

    int block_index = 0;
    int bpd = box->blocks_per_dim;

    for (int y = 0; y < bpd; y++) {
        for (int z = 0; z < bpd; z++) {
            for (int x = 0; x < bpd; x++) {
                if (block_index == MAX_RAY_BLOCKS)
                    break;

                int index = z * bpd * bpd + y * bpd + x;


                if (hitsBlock(&box->blocks[index], focalpoint)) {
                    block_indexes[block_index] = index;

                    block_index++;
                    
                        
                }
            }
        }
    }

    
	for (int i = 1; i < block_index; i++) {		// I think only 1 swap will be necessary...
		float d1_sq = (box->blocks[block_indexes[i - 1]].center - focalpoint).lenSquared();
		float d2_sq = (box->blocks[block_indexes[i]].center - focalpoint).lenSquared();

		if (d2_sq < d1_sq) {
			int tmp = block_indexes[i - 1];
			block_indexes[i - 1] = block_indexes[i];
			block_indexes[i] = tmp;
		}
	}
}

__device__ float cudaMax(float a, float b) { 
    if (a > b)
        return a;
    return b;
}
__device__ float cudaMin(float a, float b) {
    if (a < b)
        return a;
    return b;
}

__device__ bool containedBetweenPlanes(float plane_min, float plane_max, float dir_dim, float origin_dim) {
    float tmin = -999999;
    float tmax = 999999;

    float invD = 1.0f / dir_dim;
    float t0 = (plane_min - origin_dim) * invD;
    float t1 = (plane_max - origin_dim) * invD;

    if (invD < 0.0f) {
        float temp = t1;
        t1 = t0;
        t0 = temp;
    }

    tmin = t0 > tmin ? t0 : tmin;
    tmax = t1 < tmax ? t1 : tmax;



    if (tmax < tmin)    // was <=
        return false;

    return true;
}

__device__ bool Ray::hitsBlock(Block* block, Float3 focalpoint) {
    float a = BLOCK_LEN;               // FUCK YOU VISUAL STUDIO
    Float3 blocksize = Float3(a, a, a);

    Float3 offset(BODY_RADIUS, BODY_RADIUS, BODY_RADIUS);   // Radius of largest molecule!
    Float3 min = block->center - Float3(FOCUS_LEN_HALF, FOCUS_LEN_HALF, FOCUS_LEN_HALF) - offset *2;    //I have no idea why we need *2
    Float3 max = block->center + Float3(FOCUS_LEN_HALF, FOCUS_LEN_HALF, FOCUS_LEN_HALF) + offset *2;

    float tmin = -DBL_MAX;
    float tmax = DBL_MAX;

    for (int dim = 0; dim < 3; dim++) {
        float invD = unit_vector.at(dim) != 0 ? 1.0f / unit_vector.at(dim) : 999999999;

        float t0 = (min.at(dim) - focalpoint.at(dim)) * invD;
        float t1 = (max.at(dim) - focalpoint.at(dim)) * invD;

        if (invD < 0.0f) {
            float temp = t1;
            t1 = t0;
            t0 = temp;
        }

        tmin = t0 > tmin ? t0 : tmin;
        tmax = t1 < tmax ? t1 : tmax;


        if (tmax <= tmin) {    // was <=
            return false;
        }
    } 

        
    return true;

}
    

__device__ bool Ray::hitsBody(SimBody* body) {
    if (distToPoint(body->pos) < BODY_RADIUS)
        return true;
    return false;
}

__device__ bool Ray::moleculeCollisionHandling(SimBody* body, MoleculeLibrary* mol_library, uint8_t* image) {
    Molecule* mol = &mol_library->molecules[0];

    const int infinity = 9999999;

    float closest_collision = infinity;
    Atom closest_atom;
    closest_atom.pos = Float3(0, infinity, 0);  // Make sure its infinitely far away in y direction.




    Float3 molecule_tilt_vector = Float3(0, 1, 0).rotateAroundOrigin(body->rotation);
    for (int atom_index = 0; atom_index < mol->n_atoms; atom_index++) {
        
        // Local copy which we can manipulate
        Atom atom = mol->atoms[atom_index];
        atom.pos = atom.pos.rotateAroundVector(body->rotation, molecule_tilt_vector);   // Rotate around its relative origin, before moving pos to global coords
        atom.pos = atom.pos + body->pos;

        Float3 atom_absolute_pos = body->pos + atom.pos;


        if (distToPoint(atom.pos) < atom.radius) {
            float collision_dist = distToSphereIntersect(&atom);
            if (collision_dist < closest_collision) {
                closest_atom = atom;
                closest_collision = collision_dist;
            }


        }
            
    }

    if (closest_atom.pos.y != infinity) {
        int index = blockIdx.x * blockDim.x + threadIdx.x;

        image[index * 4 + 0] = closest_atom.color[0];
        image[index * 4 + 1] = closest_atom.color[1];
        image[index * 4 + 2] = closest_atom.color[2];
        image[index * 4 + 3] = 255;
        return true;
    }
    
    
    return false;
}

__device__ float Ray::distToSphereIntersect(Atom* atom) {
    Float3 projection_on_ray = origin + unit_vector * ((atom->pos - origin).dot(unit_vector) / unit_vector.dot(unit_vector));
    float center_to_projection = (projection_on_ray - atom->pos).len();
    float projection_to_intersect = sqrtf(atom->radius * atom->radius - center_to_projection * center_to_projection);
    return (projection_on_ray - origin).len() - projection_to_intersect;
}

__global__ void initRayKernel(Ray* rayptr, Box* box, Float3 focalpoint) {
	int  index = blockIdx.x * blockDim.x + threadIdx.x;
    Ray ray = rayptr[index];

	ray.findBlockHits(box, focalpoint);

    rayptr[index] = ray;
}


    

        
        
        








    





Raytracer::Raytracer(Simulation* simulation, bool verbose) {
    setGPU();


    float base = -(BOX_LEN) / 2.f;
	float principal_point_increment = (BOX_LEN) / (float)RAYS_PER_DIM;

	Ray* host_rayptr = new Ray[NUM_RAYS];
	focalpoint = Float3(0, -(BOX_LEN / 2.f) * FOCAL_LEN_RATIO - BOX_LEN, 0);

    

	int index = 0;
    for (int z_index = 0; z_index < RAYS_PER_DIM; z_index++) {
        for (int x_index = 0; x_index < RAYS_PER_DIM; x_index++) {
            float z = base + principal_point_increment * (float)z_index;
            float x = base + principal_point_increment * (float)x_index;
			Float3 vector = Float3(x, base, z) - focalpoint;
            host_rayptr[index++] = Ray(vector.norm(), focalpoint);
		}
	}

    cuda_status = hipMallocManaged(&rayptr, NUM_RAYS * sizeof(Ray));
    cuda_status = hipMemcpy(rayptr, host_rayptr, NUM_RAYS * sizeof(Ray), hipMemcpyHostToDevice);

    if (verbose) {
        printf("\n\nFocal point: %.3f %.3f %.3f\n", focalpoint.x, focalpoint.y, focalpoint.z);
        printf("Allocating %d MB of ram for Rays... \n", NUM_RAYS * sizeof(Ray) / 1000000);
    }
        

    initRayKernel <<< RAYS_PER_DIM, RAYS_PER_DIM>>> (rayptr, simulation->box, focalpoint);
    hipDeviceSynchronize();

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "rayptr init kernel failed!");
        exit(1);
    }
        
        
    if (verbose) {
        printf("Ray %d: %f %f %f\n", INDEXA, rayptr[INDEXA].unit_vector.x, rayptr[INDEXA].unit_vector.y, rayptr[INDEXA].unit_vector.z);
        printf("block_indexes: ");
        for (int i = 0; i < MAX_RAY_BLOCKS; i++) {
            printf("%d ", rayptr[INDEXA].block_indexes[i]);
        }
    }

    printf("Rays initiated\n\n");
}
    
__device__ void colorRay(Ray* ray, uint8_t* image, int index) {

}

__global__ void renderKernel(Ray* rayptr, uint8_t* image, Box* box, MoleculeLibrary* mol_library) {
    int  index = blockIdx.x * blockDim.x + threadIdx.x;
    Ray ray = rayptr[index];
    


    for (int i = 0; i < MAX_RAY_BLOCKS; i++) {
        if (ray.block_indexes[i] == -1)
            break;

        Block* block = &box->blocks[ray.block_indexes[i]];
        for (int j = 0; j < MAX_FOCUS_BODIES; j++) {

            if (block->focus_bodies[j].molecule_type == UNUSED_BODY) {  // We can do this because bodies are loaded from index 0 at each timestep. MIGHT NEED TO CHANGE IN THE FUTURE!
                break;
            }

            if (ray.hitsBody(&block->focus_bodies[j])) {
                if (ray.moleculeCollisionHandling(&block->focus_bodies[j], mol_library, image)) {
                    return;
                }
            }
        }
    }
}
    
    
uint8_t* Raytracer::render(Simulation* simulation) {
    auto start = std::chrono::high_resolution_clock::now();

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Something is wrong");
        exit(1);
    }



    hipStream_t renderstream;
    hipStreamCreate(&renderstream);


    uint8_t* cuda_image;
    int im_bytesize = NUM_RAYS * 4 * sizeof(uint8_t);
    hipMallocManaged(&cuda_image, im_bytesize);


    renderKernel << < RAYS_PER_DIM, RAYS_PER_DIM, 0>>> ( rayptr, cuda_image, simulation->box, simulation->mol_library);
    uint8_t* image = new uint8_t[NUM_RAYS * 4];
    hipMemcpy(image, cuda_image, im_bytesize, hipMemcpyDeviceToHost);


    hipFree(cuda_image);
    hipStreamDestroy(renderstream);

    auto stop = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
    printf("\tRender time: %4d ms  ", duration.count());
    // First render: 666 ms

    return image;
}