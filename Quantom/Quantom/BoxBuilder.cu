#include "hip/hip_runtime.h"
#include "BoxBuilder.cuh"



void BoxBuilder::build(Simulation* simulation) {
	simulation->box->compounds = new Compound[MAX_COMPOUNDS];
	//simulation->box->solvents = new Compound[MAX_COMPOUNDS];
	//compoundneighborlists_host = new CompoundNeighborList[MAX_COMPOUNDS];
	//compoundstates_host = new CompoundState[MAX_COMPOUNDS];


	simulation->box->compounds = new Compound[MAX_COMPOUNDS];
	simulation->box->solvents = new Solvent[MAX_SOLVENTS];

	simulation->box->compound_state_array = new CompoundState[MAX_COMPOUNDS];
	hipMalloc(&simulation->box->compound_state_array_next, sizeof(CompoundState) * MAX_COMPOUNDS);
	hipMalloc(&simulation->box->solvents_next, sizeof(Solvent) * MAX_SOLVENTS);

	simulation->box->solvent_neighborlist_array = new SolventNeighborList[MAX_COMPOUNDS + MAX_SOLVENTS];	// These are divided since both compounds and solvents will be near many more solvents than compounds
	simulation->box->compound_neighborlist_array = new CompoundNeighborList[MAX_COMPOUNDS + MAX_SOLVENTS];



	//hipMalloc(&simulation->box->compound_state_array, sizeof(CompoundState) * MAX_COMPOUNDS);
	//hipMalloc(&simulation->box->compound_state_array_next, sizeof(CompoundState) * MAX_COMPOUNDS);
	//hipMalloc(&simulation->box->compound_neighborlist_array, sizeof(CompoundNeighborList) * MAX_COMPOUNDS);

	//hipMalloc(&simulation->box->solvents, sizeof(Solvent) * MAX_SOLVENTS);

	if (N_SOLVATE_MOLECULES > 256) {
		printf("Critical indexing failure\n");
		exit(1);
	}




	placeMainMolecule(simulation);
	solvateBox(simulation);	// Always do after placing compounds
	simulation->box->total_particles = simulation->box->n_compounds * PARTICLES_PER_COMPOUND + simulation->box->n_solvents;

	compoundLinker(simulation);
	solvateLinker(simulation);
	solvateCompoundCrosslinker(simulation);



	




	Molecule water;
	for (int i = 0; i < water.n_atoms; i++) {
		simulation->box->rendermolecule.radii[i] = water.atoms[i].radius;
		for (int j = 0; j < 3; j++)
			simulation->box->rendermolecule.colors[i][j] = water.atoms[i].color[j];
	}

	simulation->box->dt = simulation->dt;






	int n_points = simulation->box->total_particles * simulation->n_steps;
	hipMalloc(&simulation->box->potE_buffer, sizeof(double) * n_points);	// Can only log molecules of size 3 for now...
	hipMalloc(&simulation->box->trajectory, sizeof(Float3) * n_points);
	//hipMemset(&simulation->box->trajectory, 0, sizeof(double) * traj_points * 3);	// uhhhhhhhhhhhhhhhhahahaha dunno bout this
	hipMallocManaged(&simulation->box->outdata, sizeof(double) * 10 * simulation->n_steps);	// 10 data streams for 10k steps. 1 step at a time.
	// 
	// 
	//hipMalloc(&simulation->box->trajectory, sizeof(Float3) * simulation->box->n_compounds * 3 * simulation->n_steps);

	simulation->box->moveToDevice();
}


void BoxBuilder::placeMainMolecule(Simulation* simulation) {
	Float3 compound_center = Float3(BOX_LEN_HALF, BOX_LEN_HALF, BOX_LEN_HALF);
	double compound_radius = 0.2;

	simulation->box->compounds[simulation->box->n_compounds++] = createCompound(
		compound_center,
		simulation->box->n_compounds,
		&simulation->box->compound_state_array[simulation->box->n_compounds],
		&simulation->box->compound_neighborlist_array[simulation->box->n_compounds],
		simulation->dt
	);
}

int BoxBuilder::solvateBox(Simulation* simulation)
{
	simulation->box->solvents = new Solvent[MAX_SOLVENTS];

	


	int bodies_per_dim = ceil(cbrt((double)N_SOLVATE_MOLECULES));
	double dist_between_compounds = (BOX_LEN) / (double)bodies_per_dim;	// dist_per_index
	double base = box_base + dist_between_compounds / 2.f;
	printf("Bodies per dim: %d. Dist per dim: %.3f\n", bodies_per_dim, dist_between_compounds);


	for (int z_index = 0; z_index < bodies_per_dim; z_index++) {
		for (int y_index = 0; y_index < bodies_per_dim; y_index++) {
			for (int x_index = 0; x_index < bodies_per_dim; x_index++) {
				if (simulation->box->n_solvents == N_SOLVATE_MOLECULES)
					break;

				Float3 solvent_center = Float3(base + dist_between_compounds * (double)x_index, base + dist_between_compounds * (double)y_index, base + dist_between_compounds * (double)z_index);
				double solvent_radius = 0.2;

				if (spaceAvailable(simulation->box, solvent_center, solvent_radius)) {
					simulation->box->solvents[simulation->box->n_solvents++] = createSolvent(
						solvent_center,
						simulation->dt
					);
				}
			}
		}
	}
	printf("%d solvents added to box\n", simulation->box->n_solvents);
	return simulation->box->n_solvents;
}







Compound BoxBuilder::createCompound(Float3 com, int compound_index, CompoundState* statebuffer_node, CompoundNeighborList* neighborinfo_node, double dt) {

	int n_atoms = PARTICLES_PER_COMPOUND;
	Float3 offsets[3] = { Float3(0,0,0), Float3(0.13, 0, 0), Float3(0, 0, -0.13) };
	for (int i = 0; i < n_atoms; i++) {
		statebuffer_node->positions[i] = com + offsets[i];	// PLACE EACH PARTICLE IN COMPOUNDS STATE, BEFORE CREATING COMPOUNDS, LETS US IMMEDIATELY CALCULATE THE COMPOUNDS CENTER OF MASS.
		statebuffer_node->n_particles++;
	}
	
	//double vrms = 250;
	Float3 compound_united_vel = Float3(v_rms , 0,0);
	Compound compound(compound_index, statebuffer_node);
	for (int i = 0; i < n_atoms; i++) {
		Float3 atom_pos_sub1 = statebuffer_node->positions[i] - compound_united_vel * dt;
		compound.particles[i] = CompactParticle(12.0107*1e-3, atom_pos_sub1);
		compound.n_particles++;
	}
	return compound;
}



Solvent BoxBuilder::createSolvent(Float3 com, double dt)	// Nodes obv. points to addresses in device global memory.
{
	Float3 solvent_vel = Float3(random(), random(), random()).norm() * v_rms;
	return 	Solvent(com, com - solvent_vel * dt);
}

bool BoxBuilder::spaceAvailable(Box* box, Float3 com, double radius) {	// Too lazy to implement yet..
	for (int i = 0; i < box->n_compounds; i++) {
		for (int j = 0; j < box->compounds[i].n_particles; j++) {
			double dist = (box->compound_state_array[i].positions[j] - com).len();
			if (dist < radius)
				return false;
		}
	}
	return true;
}





void BoxBuilder::compoundLinker(Simulation* simulation) {
	for (int i = 0; i < simulation->box->n_compounds; i++) {
		for (int j = 0; j < simulation->box->n_compounds; j++) {
			if (i != j) {
				simulation->box->compound_neighborlist_array[i].addIndex(j);
				//CompoundNeighborList* nlist = &compoundneighborlists_host[i];
				//CompoundNeighborList* nlist = &simulation->box->compound_neighborlist_array[i];
				//nlist->neighborcompound_indexes[nlist->n_neighbors++] = j;
			}
		}
	}
}

void BoxBuilder::solvateLinker(Simulation* simulation)
{
	for (int i = 0; i < simulation->box->n_solvents; i++) {
		Solvent* self = &simulation->box->solvents[i];
		for (int j = i; j < simulation->box->n_solvents; j++) {
			Solvent* other = &simulation->box->solvents[j];
			if (i != j) {
				if ((self->pos - other->pos).len() < CUTOFF) {
					simulation->box->solvent_neighborlist_array[i + MAX_COMPOUNDS].addIndex(j);
					simulation->box->solvent_neighborlist_array[j + MAX_COMPOUNDS].addIndex(i);
					//self->addNeighbor(j);		// WRONG INDEX
					//other->addNeighbor(i);		// WRONG INDEX
				}
			}
		}
	}
}

void BoxBuilder::solvateCompoundCrosslinker(Simulation* simulation)
{
	for (int i = 0; i < simulation->box->n_compounds; i++) {
		Compound* compound = &simulation->box->compounds[i];
		for (int j = 0; j < simulation->box->n_solvents; j++) {
			Solvent* solvent= &simulation->box->solvents[j];
			if ((compound->center_of_mass - solvent->pos).len() < CUTOFF) {
				simulation->box->solvent_neighborlist_array[i].addIndex(j);
				simulation->box->compound_neighborlist_array[j + MAX_COMPOUNDS].addIndex(i);

				//CompoundNeighborList* nlist = &compoundneighborlists_host[i];
				//nlist->neighborcompound_indexes[nlist->n_neighbors++] = j;
			}
		}
	}
	printf("Compound 0 solvents: %d\n", simulation->box->solvent_neighborlist_array[0].n_neighbors);
}











/*
int BoxBuilder::solvateBox(Simulation* simulation)
{
	int bodies_per_dim = ceil(cbrt((double)N_SOLVATE_MOLECULES));
	double dist_between_compounds = (BOX_LEN) / (double)bodies_per_dim;	// dist_per_index
	double base = box_base + dist_between_compounds / 2.f;
	printf("Bodies per dim: %d. Dist per dim: %.3f\n", bodies_per_dim, dist_between_compounds);


	for (int z_index = 0; z_index < bodies_per_dim; z_index++) {
		for (int y_index = 0; y_index < bodies_per_dim; y_index++) {
			for (int x_index = 0; x_index < bodies_per_dim; x_index++) {
				if (simulation->box->n_compounds == N_SOLVATE_MOLECULES)
					break;

				Float3 compound_center = Float3(base + dist_between_compounds * (double)x_index, base + dist_between_compounds * (double)y_index, base + dist_between_compounds * (double)z_index);
				double compound_radius = 0.2;

				if (spaceAvailable(compound_center, compound_radius)) {
					simulation->box->compounds[simulation->box->n_compounds++] = createSolvent(
						compound_center,
						simulation->box->n_compounds,
						&simulation->box->compound_state_array[simulation->box->n_compounds],
						&simulation->box->compound_neighborlist_array[simulation->box->n_compounds],
						simulation->dt
					);
				}
			}
		}
	}
	return simulation->box->n_compounds;
}
*/