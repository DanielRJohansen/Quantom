#include "hip/hip_runtime.h"
#include "Analyzer.cuh"






template<typename T>
void __device__ distributedSummation(T* arrayptr, int array_len) {				// Places the result at pos 0 of input_array
	T temp;			// This is a lazy soluation, but maybe it is also fast? Definitely simple..
	for (int i = 1; i < array_len; i *= 2) {	// Distributed averaging							// Make a generic and SAFER function for this, PLEASE OK??
		if ((threadIdx.x + i) < array_len) {
			temp = arrayptr[threadIdx.x] + arrayptr[threadIdx.x + i];// *0.5f;	// easier to just divide by sum of solvents at host
		}
		__syncthreads();
		arrayptr[threadIdx.x] = temp;
		__syncthreads();
	}
}

double __device__ calcKineticEnergy(Float3* pos1, Float3* pos2, double mass, double dt) {
	LIMAENG::applyHyperpos(pos1, pos2);


	double vel = (*pos1 - *pos2).len() * 0.5f / dt;
	double kinE = 0.5 * mass * vel * vel;
	return kinE;
}

void __global__ monitorCompoundEnergyKernel(Box* box, Float3* traj_buffer, double* potE_buffer, Float3* data_out) {		// everything here breaks if not all compounds are identical in particle count and particle mass!!!!!!!
	__shared__ Float3 energy[MAX_COMPOUND_PARTICLES];
	__shared__ Compound compound;


	const int step = blockIdx.x + 1;
	const int compound_index = blockIdx.y;
	energy[threadIdx.x] = Float3(0.f);

	if (threadIdx.x == 0) {
		//printf("index: %d\n", compound_index + (step - 1) * N_MONITORBLOCKS_PER_STEP);
		data_out[compound_index + (step - 1) * box->n_compounds] = Float3(0, 0, 0);
		//mass = box->compounds[compound_index].particles[0]
		compound = box->compounds[compound_index];
	}

	__syncthreads();

	if (threadIdx.x >= compound.n_particles) {
		return;
	}
	__syncthreads();


	double potE = potE_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + step * box->total_particles_upperbound];

	Float3 pos_tsub1 = traj_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + (step - 1) * box->total_particles_upperbound];
	Float3 pos_tadd1 = traj_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + (step + 1) * box->total_particles_upperbound];
	//LIMAENG::applyHyperpos(&pos_tadd1, &pos_tsub1);
	//testspace::testerfn(4);
	//double kinE = calcKineticEnergy(&pos_tadd1, &pos_tsub1, compound.particles[threadIdx.x].mass, box->dt);
	double kinE = calcKineticEnergy(&pos_tadd1, &pos_tsub1, SOLVENT_MASS, box->dt);
	//double kinE = calcKineticEnergy(&pos_tadd1, &pos_tsub1, forcefield_device, box->dt);
	/*
	applyHyperposA(&pos_tadd1, &pos_tsub1);
	double vel = (pos_tadd1 - pos_tsub1).len() * 0.5f / box->dt;
	double kinE = 0.5 * compound.particles[threadIdx.x].mass * vel * vel;
	*/


	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();


	/*if (energy[threadIdx.x].len() > 40000) {
		printf("Step %d\n", step);
		energy[threadIdx.x].print('c');
	}*/

	distributedSummation(energy, MAX_COMPOUND_PARTICLES);
	
	__syncthreads();
	if (threadIdx.x == 0) {
		data_out[compound_index + (step - 1) * box->n_compounds] = energy[0];
	}
}





void __global__ monitorSolventEnergyKernel(Box* box, Float3* traj_buffer, double* potE_buffer, Float3* data_out) {
	__shared__ Float3 energy[THREADS_PER_SOLVENTBLOCK];



	int solvent_index = threadIdx.x + blockIdx.y * THREADS_PER_SOLVENTBLOCK;
	int step = blockIdx.x + 1;
	int compounds_offset = box->n_compounds * MAX_COMPOUND_PARTICLES;


	energy[threadIdx.x] = Float3(0.f);

	if (threadIdx.x == 0) {
		//data_out[blockIdx.y + (step - 1) * N_MONITORBLOCKS_PER_STEP] = Float3(0, 0, 0);
	}
	if (solvent_index >= box->n_solvents) {	// Shouldn't be necessary right now..
		return;
	}



	Float3 pos_tsub1 = traj_buffer[compounds_offset + solvent_index + (step - 1) * box->total_particles_upperbound];
	Float3 pos_tadd1 = traj_buffer[compounds_offset + solvent_index + (step + 1) * box->total_particles_upperbound];
	double kinE = calcKineticEnergy(&pos_tadd1, &pos_tsub1, SOLVENT_MASS, box->dt);

	double potE = potE_buffer[compounds_offset + solvent_index + step * box->total_particles_upperbound];

	if (potE > 200'000) {
		//printf("step %04d solvate %04d pot %f\n", step, solvent_index, potE);
	}

	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);

	__syncthreads();

	distributedSummation(energy, THREADS_PER_SOLVENTBLOCK);
	if (threadIdx.x == 0) {
		data_out[(step-1) * gridDim.y + blockIdx.y] = energy[0];
	}
}



Analyzer::AnalyzedPackage Analyzer::analyzeEnergy(Simulation* simulation) {	// Calculates the avg J/mol // calculate energies separately for compounds and solvents. weigh averages based on amount of each
	int analysable_steps = simulation->getStep() - 3;
	if (analysable_steps < 1) {
		printf("FATAL ERROR, no steps to analyze");
		exit(1);
	}
		

	Float3* average_energy = new Float3[analysable_steps];


																		// TODO: I think maybe i am missing 1 datapoint here? Something about only loading 99 steps in??
	hipMalloc(&traj_buffer_device, sizeof(Float3) * simulation->total_particles_upperbound * simulation->getStep());
	hipMalloc(&potE_buffer_device, sizeof(double) * simulation->total_particles_upperbound * simulation->getStep());
	hipMemcpy(traj_buffer_device, simulation->traj_buffer, sizeof(Float3) * simulation->total_particles_upperbound * simulation->getStep(), hipMemcpyHostToDevice);
	hipMemcpy(potE_buffer_device, simulation->potE_buffer, sizeof(double) * simulation->total_particles_upperbound * simulation->getStep(), hipMemcpyHostToDevice);


	Float3* average_solvent_energy = analyzeSolvateEnergy(simulation, analysable_steps);
	Float3* average_compound_energy = analyzeCompoundEnergy(simulation, analysable_steps);	//avg energy PER PARTICLE in compound

	for (int i = 0; i < analysable_steps; i++) {
		average_energy[i] = (average_solvent_energy[i] * simulation->box->n_solvents * 1 + average_compound_energy[i] * simulation->box->compounds[0].n_particles) * (1.f/ (simulation->box->n_solvents + simulation->box->compounds[0].n_particles));
		//average_energy[i].print('E');
	}
	

	hipFree(traj_buffer_device);
	hipFree(potE_buffer_device);
	delete [] average_solvent_energy, average_compound_energy;


	printf("\n########## Finished analyzing energies ##########\n\n");
	return AnalyzedPackage(average_energy, analysable_steps, simulation->temperature_buffer, simulation->getStep() / STEPS_PER_THERMOSTAT);;
}

Float3* Analyzer::analyzeSolvateEnergy(Simulation* simulation, int n_steps) {
	dim3 block_dim(n_steps, BLOCKS_PER_SOLVENTKERNEL, 1);
	Float3* average_solvent_energy = new Float3[n_steps];
	Float3* average_solvent_energy_blocked = new Float3[n_steps * BLOCKS_PER_SOLVENTKERNEL];
	Float3* data_out;
	hipMalloc(&data_out, sizeof(Float3) * BLOCKS_PER_SOLVENTKERNEL * n_steps);


	//printf("Fix this to do analyzing >256\n");
	//exit(1);
	monitorSolventEnergyKernel << < block_dim, THREADS_PER_SOLVENTBLOCK >> > (simulation->box, traj_buffer_device, potE_buffer_device, data_out);
	hipDeviceSynchronize();
	hipMemcpy(average_solvent_energy_blocked, data_out, sizeof(Float3) * BLOCKS_PER_SOLVENTKERNEL * (n_steps), hipMemcpyDeviceToHost);

	for (int step = 0; step < n_steps; step++) {
		average_solvent_energy[step] = Float3(0.f);
		for (int block = 0; block < BLOCKS_PER_SOLVENTKERNEL; block++) {
			average_solvent_energy[step] += average_solvent_energy_blocked[block + step * BLOCKS_PER_SOLVENTKERNEL];
		}
		average_solvent_energy[step] *= (1.f / simulation->n_solvents);
	}


	hipFree(data_out);
	delete[] average_solvent_energy_blocked;
	LIMAENG::genericErrorCheck("Cuda error during analyzeSolvateEnergy\n");

	return average_solvent_energy;
}


Float3* Analyzer::analyzeCompoundEnergy(Simulation* simulation, int n_steps) {
	int n_datapoints = simulation->n_compounds * n_steps;
	printf("n steps: %d\n", n_steps);
	dim3 block_dim(n_steps, simulation->box->n_compounds, 1);
	Float3* average_compound_energy = new Float3[n_steps];
	Float3* host_data = new Float3[n_datapoints];
	Float3* data_out;
	hipMalloc(&data_out, sizeof(Float3) * n_datapoints);




	monitorCompoundEnergyKernel << < block_dim, MAX_COMPOUND_PARTICLES >> > (simulation->box, traj_buffer_device, potE_buffer_device, data_out);
	hipDeviceSynchronize();
	hipMemcpy(host_data, data_out, sizeof(Float3) * n_datapoints, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	printf("This energy analysis is only valid if just 1 compound exists!\n");
	for (int step = 0; step < n_steps; step++) {
		for (int i = 0; i < simulation->box->n_compounds; i++) {
			//if (host_data[i + step * 256].x > 10000)
				//printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_compound_energy[step] += host_data[i + step * simulation->box->n_compounds];
		}
		average_compound_energy[step] *= (1.f / (simulation->box->compounds[0].n_particles));
	}


	hipFree(data_out);
	delete[] host_data;
	LIMAENG::genericErrorCheck("Cuda error during analyzeCompoundEnergy\n");

	return average_compound_energy;
}















/*
void Analyzer::printEnergies(Float3* energy_data, int analysable_steps, Simulation* simulation) {
	string file_path_s = "D:\\Quantom\\energies_steps_" + to_string(analysable_steps) + ".bin";
	char* file_path;
	file_path = &file_path_s[0];
	cout << "Printing to file " << file_path << endl;

	FILE* file;
	fopen_s(&file, file_path, "wb");
	fwrite(energy_data, sizeof(Float3), analysable_steps, file);
	fclose(file);



	file_path_s = "D:\\Quantom\\temperatures_steps_" + to_string(analysable_steps) + ".bin";
	file_path = &file_path_s[0];
	cout << "Printing to file " << file_path << endl;

	fopen_s(&file, file_path, "wb");
	fwrite(simulation->temperature_buffer, sizeof(float), simulation->getStep()/STEPS_PER_THERMOSTAT, file);
	fclose(file);
}



*/