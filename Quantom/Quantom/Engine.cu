#include "hip/hip_runtime.h"
#include "Engine.cuh"




Simulation* Engine::prepSimulation(Simulation* simulation) {
	this->simulation = simulation;
	simulation->bodies = new SimBody[simulation->n_bodies];


	int n_blocks = initBlocks();
	//linkBlocks();
	//prepareEdgeBlocks();

	srand(290128301);
	int n_bodies = fillBox();


	printf("\nSimbody size: %d bytes\n", sizeof(SimBody));
	printf("Block size: %d\n", sizeof(Block));
	printf("Simulation configured with %d blocks, and %d bodies. Approximately %d bodies per block. \n", n_blocks, n_bodies, n_bodies/n_blocks);
	printf("Required shared mem for stepKernel: %d\n", sizeof(Block));
	printf("Required global mem for Box: %d MB\n", (int) (sizeof(Block) * n_blocks / 1000000.f));
	//exit(1);


	prepareCudaScheduler();

	return simToDevice();
}

Simulation* Engine::simToDevice() {
	simulation->moveToDevice();	// Must be done before initiating raytracer!

	Simulation* temp;
	int bytesize = sizeof(Simulation);
	hipMallocManaged(&temp, bytesize);
	hipMemcpy(temp, simulation, bytesize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	delete simulation;
	simulation = temp;

	return simulation;
}

int Engine::initBlocks() {
	

	block_dist = FOCUS_LEN;
	bpd = ((simulation->box_size + FOCUS_LEN)/ block_dist);		// Otherwise no block focus on edge particles
	int n_blocks = pow(bpd, 3);
	box_base = - (BOX_LEN/2.f);			// Is the center of first block!
	//float block_center_base = box_base + block_dist;


	printf("Blocks per dim: %d\n", bpd);

	simulation->blocks_per_dim = bpd;
	simulation->box->blocks_per_dim = bpd;
	simulation->box->n_blocks = n_blocks;
	simulation->box->blocks = new Block[n_blocks];


	int index = 0;
	float offset = -simulation->box_size / 2 + 0.5 * BLOCK_LEN;
	for (int x = 0; x < bpd; x++) {
		for (int y = 0; y < bpd; y++) {
			for (int z = 0; z < bpd; z++) {
				Float3 center(x * block_dist + box_base, y * block_dist + box_base, z * block_dist + box_base);
				//center.print();
				simulation->box->blocks[index++] = Block(center);
			}
		}
	}
	return index;
}

int Engine::fillBox() {
	int bodies_per_dim = ceil(cbrt((float)simulation->n_bodies));
	printf("Bodies per dim: %d\n", bodies_per_dim);
	float dist = simulation->box_size / (float)bodies_per_dim;	// dist_per_index
	float base = -simulation->box_size / 2.f + dist / 2.f;

	float vel_scalar = 0.5;

	int index = 0;
	for (int x_index = 0; x_index < bodies_per_dim; x_index++) {
		for (int y_index = 0; y_index < bodies_per_dim; y_index++) {
			for (int z_index = 0; z_index < bodies_per_dim; z_index++) {
				if (index == simulation->n_bodies)
					break;

				int p = 10000;
				float r1 = rand() % p / (float)p - 0.5;
				float r2 = rand() % p / (float)p - 0.5;
				float r3 = rand() % p / (float)p - 0.5;

				simulation->bodies[index].pos = Float3(base + dist * (float)x_index, base + dist * float(y_index), base + dist * float(z_index));

				//printf("Body pos: ");
				//simulation->bodies[index].pos.print();

				simulation->bodies[index].molecule_type = 0;
				simulation->bodies[index].vel = Float3(r1 * vel_scalar, r2 * vel_scalar, r3 * vel_scalar);
				simulation->bodies[index].rotation = Float3(0, 0, 0);
				simulation->bodies[index].rot_vel = Float3(0, PI, 0);
				placeBody(&simulation->bodies[index++]);
			}
		}
	}
	return index;
}
/*
void Engine::placeBody(SimBody* body) {
	//const Int3 block_index = posToBlockIndex(&body->pos);
	

	int count = 0;

	SimBody temp;
	Int3 block_index_;

	for (int z_off = -1; z_off <= 1; z_off++) {
		for (int y_off = -1; y_off <= 1; y_off++) {
			for (int x_off = -1; x_off <= 1; x_off++) {


				Float3 pos_(body->pos.x + x_off * BLOCK_OVERLAP, body->pos.y + y_off * BLOCK_OVERLAP, body->pos.z + z_off * BLOCK_OVERLAP);
				block_index_ = posToBlockIndex(&pos_);
				//printf("%d %d %d\n", block_index_.x, block_index_.y, block_index_.z);
				int block_index_1d = block3dIndexTo1dIndex(block_index_);
				//printf("Block index. %d\n", block_index_1d);
				Block* block = &simulation->box->blocks[block_index_1d];
				if (block->addBody(body))
					count++;
			}
		}
	}
	//printf("Molecule placed in %d blocks\n", count);
}
*/

void Engine::placeBody(SimBody* body) {
	Int3 block_index = posToBlockIndex(&body->pos);
	//printf("Block index: %d %d %d\n", block_index.x, block_index.y, block_index.z);

	int block_index_1d = block3dIndexTo1dIndex(block_index);
	//printf("Block index: %d\n", block_index_1d);

	if (block_index_1d < 0)
		printf("Rebuild All you twat\n");

	Block* block = &simulation->box->blocks[block_index_1d];

	if (!block->addBody(body))
		printf("Body lost!\n");

}
	




void Engine::prepareCudaScheduler() {
	sim_blocks = simulation->box->n_blocks;

	for (int i = 0; i < N_STREAMS; i++)
		hipStreamCreate(&stream[i]);

	printf("%d kernel launches necessary to step\n", (int) ceil((float)simulation->box->n_blocks / (float)BLOCKS_PER_SM));
	//gridblock_size = dim3(GRIDBLOCKS_PER_BODY, BLOCKS_PER_SM, 1);
}


		

	


//--------------------------------------------------------------------------	SIMULATION BEGINS HERE --------------------------------------------------------------//


void Engine::step() {
	auto start = std::chrono::high_resolution_clock::now();

	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "Error before step!");
		exit(1);
	}




	int blocks_handled = 0;
	int sharedmem_size = sizeof(Block);
	while (blocks_handled < sim_blocks) {
		for (int i = 0; i < N_STREAMS; i++) {
			stepKernel << < BLOCKS_PER_SM, MAX_FOCUS_BODIES, 0, stream[i] >> > (simulation, blocks_handled);

			

			blocks_handled += BLOCKS_PER_SM;

			if (blocks_handled >= sim_blocks)
				break;
		}

		hipDeviceSynchronize();
		if (hipGetLastError() != hipSuccess) {
			fprintf(stderr, "Error during step :/\n");
			exit(1);
		}

	}
	


	/*
	stepKernel <<< sim_blocks, MAX_BLOCK_BODIES, sizeof(int) >> > (simulation);
	hipDeviceSynchronize();
*/

	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	printf("Step time: %d ys", duration.count());
}

__device__ float cudaMax1(float a, float b) {
	if (a > b)
		return a;
	return b;
}
__device__ float cudaMin1(float a, float b) {
	if (a < b)
		return a;
	return b;
}

__device__ Float3 forceFromDist(Float3 dists) {
	return Float3(
		((0.5f / dists.x) + 0.5),
		((0.5f / dists.y) + 0.5),
		((0.5f / dists.z) + 0.5)
	);
}

__device__ inline Float3 calcEdgeForce(Block* block, SimBody* body) {
	Float3 dists1(body->pos.x + BOX_LEN / 2.f, body->pos.y + BOX_LEN / 2.f, body->pos.z + BOX_LEN / 2.f);
	Float3 dists2(BOX_LEN / 2.f - body->pos.x, BOX_LEN / 2.f - body->pos.y, BOX_LEN / 2.f - body->pos.z);


	Float3 pos_forces = forceFromDist(dists1).zeroIfBelow(0);
	Float3 neg_forces = forceFromDist(dists2).zeroIfBelow(0);
	return pos_forces - neg_forces;
}


enum Direction
{
	up, down, left, right, back, forward
};

__device__ void transferBody(Box* box, Block* block, SimBody* body) {
	Float3 rel_pos = body->pos - block->center;

	//switch (rel_pos.x)
}

																				// TODO: MAKE IS SUCH THAT A BODY CAN NEVER BE EXACTLY ON THE EDGE OF FOCUS, THUS APPEARING IN MULTIPLE GROUPS!
__device__ bool bodyInNear(SimBody* body, Float3* block_center) {
	Float3 dist_from_center = (body->pos - *block_center).abs();
	return (dist_from_center.x < FOCUS_LEN, dist_from_center.z < FOCUS_LEN, dist_from_center.z < FOCUS_LEN);
}

__device__ bool bodyInFocus(SimBody* body, Float3* block_center) {
	Float3 dist_from_center = (body->pos - *block_center).abs();
	return (dist_from_center.x < FOCUS_LEN_HALF, dist_from_center.z < FOCUS_LEN_HALF, dist_from_center.z < FOCUS_LEN_HALF);
}





__global__ void stepKernel(Simulation* simulation, int offset) {
	int blockID = blockIdx.x + offset;
	int bodyID = threadIdx.x;

	if (blockID >= simulation->box->n_blocks)
		return;
	




	

	


	// Load bodies into shared memory
	__shared__ Block block;	
	if (threadIdx.x == 0)
		block = simulation->box->blocks[blockID];


	int focus_ptr = 0;

	if (threadIdx.x == 0) {

		while (block.focus_bodies[focus_ptr].molecule_type != UNUSED_BODY)
			focus_ptr++;


		for (int i = 0; i < MAX_NEAR_BODIES; i++) {
			SimBody body = block.near_bodies[i];

			if (body.molecule_type == UNUSED_BODY)
				continue;				// Maybe break if always sorted??

			if (focus_ptr == 16)
				printf("FUCK! %d\n", blockID);

			if (bodyInFocus(&body, &block.center)) {
				block.focus_bodies[focus_ptr++] = body;
				block.near_bodies[i].molecule_type = UNUSED_BODY;
			}
		}
		//printf("%d focusses\n", focus_ptr);
	}
	__syncthreads();


	


	// End thread if not needed.
	if (block.focus_bodies[bodyID].molecule_type == UNUSED_BODY && bodyID != 0)		// Always need thread0 to send block global
		return;
		//printf("What the fuckj is going on. n focusses %d\n", focus_ptr);
		//return;


	// BEGIN WORK
	SimBody body = block.focus_bodies[bodyID];
	


	
	//if (block.edge_block)
		//body.vel = body.vel + calcEdgeForce(&block, &body) * 0.1;

	//if (abs(body.pos.x - block.center.x) > SOLOBLOCK_DIST) {
	//}
	


	body.rotation = body.rotation + body.rot_vel * simulation->dt;				// * dt of course!

	//body.pos = body.pos + body.vel * simulation->dt;

	block.focus_bodies[bodyID] = body;
	



	__syncthreads();
	if (bodyID == 0)
		simulation->box->blocks[blockID] = block;	// Very expensive..

	
} 