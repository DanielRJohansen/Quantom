#include "hip/hip_runtime.h"
#include "Analyzer.cuh"






void __device__ applyHyperposA(Float3* static_particle, Float3* movable_particle) {
	for (int i = 0; i < 3; i++) {
		*movable_particle->placeAt(i) += BOX_LEN * ((static_particle->at(i) - movable_particle->at(i)) > BOX_LEN_HALF);
		*movable_particle->placeAt(i) -= BOX_LEN * ((static_particle->at(i) - movable_particle->at(i)) < -BOX_LEN_HALF);	// use at not X!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	}
}


void __global__ monitorCompoundEnergyKernel(Box* box, Float3* traj_buffer, double* potE_buffer, Float3* data_out) {		// everything here breaks if not all compounds are identical in particle count and particle mass!!!!!!!
	__shared__ Float3 energy[MAX_COMPOUND_PARTICLES];
	__shared__ Compound compound;


	const int step = blockIdx.x + 1;
	const int compound_index = blockIdx.y;
	energy[threadIdx.x] = Float3(0, 0, 0);

	if (threadIdx.x == 0) {
		//printf("index: %d\n", compound_index + (step - 1) * N_MONITORBLOCKS_PER_STEP);
		data_out[compound_index + (step - 1) * box->n_compounds] = Float3(0, 0, 0);
		//mass = box->compounds[compound_index].particles[0]
		compound = box->compounds[compound_index];
	}

	__syncthreads();

	if (threadIdx.x >= compound.n_particles) {
		return;
	}
	__syncthreads();


	double potE = potE_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + step * box->total_particles_upperbound];

	Float3 pos_tsub1 = traj_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + (step - 1) * box->total_particles_upperbound];
	Float3 pos_tadd1 = traj_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + (step + 1) * box->total_particles_upperbound];
	applyHyperposA(&pos_tadd1, &pos_tsub1);
	

	double vel = (pos_tadd1 - pos_tsub1).len() * 0.5f / box->dt;
	double kinE = 0.5 * compound.particles[threadIdx.x].mass * vel * vel;

	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();



	for (int i = 1; i < MAX_COMPOUND_PARTICLES; i *= 2) {	// Distributed averaging							// Make a generic and SAFER function for this, PLEASE OK??
		Float3 temp;			// This is a lazy soluation, but maybe it is also fast? Definitely simple..
		if ((threadIdx.x + i) < MAX_COMPOUND_PARTICLES) {
			//energy[threadIdx.x] = (energy[threadIdx.x] + energy[threadIdx.x + i]);// *0.5f;	// easier to just divide by sum of solvents at host
			temp = energy[threadIdx.x] + energy[threadIdx.x + i];// *0.5f;	// easier to just divide by sum of solvents at host
		}
		__syncthreads();
		energy[threadIdx.x] = temp;
		__syncthreads();
	}


	__syncthreads();
	if (threadIdx.x == 0) {
		data_out[compound_index + (step - 1) * box->n_compounds] = energy[0];
	}
}




/*
void __global__ monitorSolventEnergyKernel(Box* box, Float3* data_out) {
	__shared__ Float3 energy[THREADS_PER_MONITORBLOCK];
	int solvent_index = threadIdx.x + blockIdx.y * THREADS_PER_MONITORBLOCK;
	int step = blockIdx.x + 1;
	int compounds_offset = box->n_compounds * PARTICLES_PER_COMPOUND;


	energy[threadIdx.x] = Float3(0, 0, 0);
	if (solvent_index >= box->n_solvents) {
		return;
	}


	if (threadIdx.x == 0) {
		data_out[blockIdx.y + (step - 1) * N_MONITORBLOCKS_PER_STEP] = Float3(0, 0, 0);
	}

	Float3 pos_tsub1 = box->trajectory[compounds_offset + solvent_index + (step - 1) * box->total_particles_upperbound];
	Float3 pos_tadd1 = box->trajectory[compounds_offset + solvent_index + (step + 1) * box->total_particles_upperbound];
	applyHyperposA(&pos_tadd1, &pos_tsub1);


	double potE = box->potE_buffer[compounds_offset + solvent_index + step * box->total_particles_upperbound];

	
	double vel = (pos_tadd1 - pos_tsub1).len() * 0.5f / box->dt;
	if (vel > 10'000) {
		printf("step %04d solvate %04dvel:  %f\n", step, solvent_index, vel);
		//pos_tadd1.print('a');
		//printf("analyzer index %d\n", compounds_offset + solvent_index + (step - 1) * box->total_particles_upperbound);
		//pos_tsub1.print('s');
	}
	if (potE > 200'000) {
		printf("step %04d solvate %04d pot %f\n", step, solvent_index, potE);
	}
	
		
	double kinE = 0.5 * SOLVENT_MASS * vel * vel;


	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();

	for (int i = 1; i < THREADS_PER_MONITORBLOCK; i *= 2) {	// Distributed averaging
		if ((threadIdx.x + i) < THREADS_PER_MONITORBLOCK) {
			energy[threadIdx.x] = (energy[threadIdx.x] + energy[threadIdx.x + i]);// *0.5f;	// easier to just divide by sum of solvents at host
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		//energy[0].print('b');
		data_out[blockIdx.y + (step-1) * N_MONITORBLOCKS_PER_STEP] = energy[0];
		//data_out[blockIdx.y + (step - 1) * 256] = Float3(1,2,3);
		//data_out[blockIdx.y + step * 256].print('g');
	}
	
}
*/


void Analyzer::analyzeEnergy(Simulation* simulation) {	// Calculates the avg J/mol // calculate energies separately for compounds and solvents. weigh averages based on amount of each
	int analysable_steps = simulation->getStep() - 2;
	if (analysable_steps < 1)
		return;

	Float3* average_energy = new Float3[analysable_steps];
	

	//Float3* average_solvent_energy = analyzeSolvateEnergy(simulation, analysable_steps);
	Float3* average_compound_energy = analyzeCompoundEnergy(simulation, analysable_steps);	//avg energy PER PARTICLE in compound

	

	for (int i = 0; i < analysable_steps; i++) {
		average_compound_energy->print('e');
		//average_energy[i] = (average_solvent_energy[i] * simulation->box->n_solvents * 1 + average_compound_energy[i] * simulation->box->n_compounds * PARTICLES_PER_COMPOUND) * (1.f/ simulation->box->total_particles_upperbound);
	}


	printEnergies(average_energy, analysable_steps);


	//delete [] average_solvent_energy, average_compound_energy, average_energy;


	printf("\n########## Finished analyzing energies ##########\n\n");

}
/*
Float3* Analyzer::analyzeSolvateEnergy(Simulation* simulation, int n_steps)
{
	// Solvent energies first //
	dim3 block_dim(n_steps, 256, 1);
	Float3* average_solvent_energy = new Float3[n_steps];
	Float3* host_data = new Float3[256 * n_steps];
	Float3* device_data;
	hipMalloc(&device_data, sizeof(Float3) * 256 * n_steps);


	monitorSolventEnergyKernel <<< block_dim, 256 >>> (simulation->box, device_data);
	hipDeviceSynchronize();
	hipMemcpy(host_data, device_data, sizeof(Float3) * 256 * (n_steps), hipMemcpyDeviceToHost);

	for (int step = 0; step < n_steps; step++) {
		for (int i = 0; i < 256; i++) {
			if (host_data[i + step * 256].x > 10000)
				printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_solvent_energy[step] += host_data[i + step * 256];
		}
		average_solvent_energy[step] *= (1.f / simulation->box->n_solvents);
	}


	hipFree(device_data);
	delete[] host_data;

	return average_solvent_energy;
}
*/
Float3* Analyzer::analyzeCompoundEnergy(Simulation* simulation, int n_steps) {
	int n_datapoints = simulation->n_compounds * n_steps;
	printf("n steps: %d\n", n_steps);
	dim3 block_dim(n_steps, simulation->box->n_compounds, 1);
	Float3* average_compound_energy = new Float3[n_steps];
	Float3* host_data = new Float3[n_datapoints];
	Float3* device_data;
	hipMalloc(&device_data, sizeof(Float3) * n_datapoints);


	Float3* traj_buffer_device;
	double* potE_buffer_device;
	hipMalloc(&traj_buffer_device, sizeof(Float3) * simulation->total_particles_upperbound * simulation->getStep());
	hipMalloc(&potE_buffer_device, sizeof(double) * simulation->total_particles_upperbound * simulation->getStep());
	hipMemcpy(traj_buffer_device, simulation->traj_buffer, sizeof(Float3) * simulation->total_particles_upperbound * simulation->getStep(), hipMemcpyHostToDevice);
	hipMemcpy(potE_buffer_device, simulation->potE_buffer, sizeof(double) * simulation->total_particles_upperbound * simulation->getStep(), hipMemcpyHostToDevice);

	monitorCompoundEnergyKernel << < block_dim, MAX_COMPOUND_PARTICLES >> > (simulation->box, traj_buffer_device, potE_buffer_device, device_data);
	hipDeviceSynchronize();
	hipMemcpy(host_data, device_data, sizeof(Float3) * n_datapoints, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	printf("What %d\n", simulation->box->n_compounds);
	for (int step = 0; step < n_steps; step++) {
		for (int i = 0; i < simulation->box->n_compounds; i++) {
			//if (host_data[i + step * 256].x > 10000)
				//printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_compound_energy[step] += host_data[i + step * simulation->box->n_compounds];
		}
		//average_compound_energy[step] *= (1.f / (simulation->box->n_compounds * PARTICLES_PER_COMPOUND));
	}


	hipFree(device_data);
	delete[] host_data;

	return average_compound_energy;
}


void Analyzer::printEnergies(Float3* energy_data, int analysable_steps) {
	std::ofstream myfile("D:\\Quantom\\energies.csv");


	for (int i = 0; i < analysable_steps; i++) {
		for (int j = 0; j < 3; j++) {
			myfile << energy_data[i].at(j) << ';';
			//myfile << energy_data[j + i * 3] << ";";
		}
		myfile << "\n";
	}
	myfile.close();
}
























