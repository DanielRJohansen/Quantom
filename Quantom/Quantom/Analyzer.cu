#include "hip/hip_runtime.h"
#include "Analyzer.cuh"






template<typename T>
void __device__ distributedSummation(T* arrayptr, int array_len) {				// Places the result at pos 0 of input_array
	T temp;			// This is a lazy soluation, but maybe it is also fast? Definitely simple..
	for (int i = 1; i < array_len; i *= 2) {	// Distributed averaging							// Make a generic and SAFER function for this, PLEASE OK??
		if ((threadIdx.x + i) < array_len) {
			temp = arrayptr[threadIdx.x] + arrayptr[threadIdx.x + i];// *0.5f;	// easier to just divide by sum of solvents at host
		}
		__syncthreads();
		arrayptr[threadIdx.x] = temp;
		__syncthreads();
	}
}

double __device__ calcKineticEnergy(Float3* pos1, Float3* pos2, double mass, double dt) {
	LIMAENG::applyHyperpos(pos1, pos2);


	double vel = (*pos1 - *pos2).len() * 0.5f / dt;
	double kinE = 0.5 * mass * vel * vel;
	return kinE;
}

void __global__ monitorCompoundEnergyKernel(Box* box, Float3* traj_buffer, double* potE_buffer, Float3* data_out) {		// everything here breaks if not all compounds are identical in particle count and particle mass!!!!!!!
	__shared__ Float3 energy[MAX_COMPOUND_PARTICLES];
	__shared__ Compound compound;


	const int step = blockIdx.x + 1;
	const int compound_index = blockIdx.y;
	energy[threadIdx.x] = Float3(0.f);

	if (threadIdx.x == 0) {
		//printf("index: %d\n", compound_index + (step - 1) * N_MONITORBLOCKS_PER_STEP);
		data_out[compound_index + (step - 1) * box->n_compounds] = Float3(0, 0, 0);
		//mass = box->compounds[compound_index].particles[0]
		compound = box->compounds[compound_index];
	}

	__syncthreads();

	if (threadIdx.x >= compound.n_particles) {
		return;
	}
	__syncthreads();


	double potE = potE_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + step * box->total_particles_upperbound];

	Float3 pos_tsub1 = traj_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + (step - 1) * box->total_particles_upperbound];
	Float3 pos_tadd1 = traj_buffer[threadIdx.x + compound_index * MAX_COMPOUND_PARTICLES + (step + 1) * box->total_particles_upperbound];
	//LIMAENG::applyHyperpos(&pos_tadd1, &pos_tsub1);
	//testspace::testerfn(4);
	double kinE = calcKineticEnergy(&pos_tadd1, &pos_tsub1, compound.particles[threadIdx.x].mass, box->dt);
	/*
	applyHyperposA(&pos_tadd1, &pos_tsub1);
	double vel = (pos_tadd1 - pos_tsub1).len() * 0.5f / box->dt;
	double kinE = 0.5 * compound.particles[threadIdx.x].mass * vel * vel;
	*/


	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();


	/*if (energy[threadIdx.x].len() > 40000) {
		printf("Step %d\n", step);
		energy[threadIdx.x].print('c');
	}*/

	distributedSummation(energy, MAX_COMPOUND_PARTICLES);
	
	__syncthreads();
	if (threadIdx.x == 0) {
		data_out[compound_index + (step - 1) * box->n_compounds] = energy[0];
	}
}





void __global__ monitorSolventEnergyKernel(Box* box, Float3* traj_buffer, double* potE_buffer, Float3* data_out) {
	__shared__ Float3 energy[256];

	//int solvent_index = threadIdx.x + blockIdx.y * THREADS_PER_MONITORBLOCK;
	int solvent_index = threadIdx.x;
	int step = blockIdx.x + 1;
	int compounds_offset = box->n_compounds * MAX_COMPOUND_PARTICLES;


	energy[threadIdx.x] = Float3(0.f);

	if (threadIdx.x == 0) {
		//data_out[blockIdx.y + (step - 1) * N_MONITORBLOCKS_PER_STEP] = Float3(0, 0, 0);
	}
	if (solvent_index >= box->n_solvents) {	// Shouldn't be necessary right now..
		return;
	}



	Float3 pos_tsub1 = traj_buffer[compounds_offset + solvent_index + (step - 1) * box->total_particles_upperbound];
	Float3 pos_tadd1 = traj_buffer[compounds_offset + solvent_index + (step + 1) * box->total_particles_upperbound];
	double kinE = calcKineticEnergy(&pos_tadd1, &pos_tsub1, SOLVENT_MASS, box->dt);

	double potE = potE_buffer[compounds_offset + solvent_index + step * box->total_particles_upperbound];

	if (potE > 200'000) {
		//printf("step %04d solvate %04d pot %f\n", step, solvent_index, potE);
	}

	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();


		

	distributedSummation(energy, 256);
	if (threadIdx.x == 0) {
		//energy[0].print('b');
		data_out[step-1] = energy[0];
		//data_out[blockIdx.y + (step - 1) * 256] = Float3(1,2,3);
		//data_out[blockIdx.y + step * 256].print('g');
	}	
}



void Analyzer::analyzeEnergy(Simulation* simulation) {	// Calculates the avg J/mol // calculate energies separately for compounds and solvents. weigh averages based on amount of each
	printf("Get step %d\n", simulation->getStep());
	int analysable_steps = simulation->getStep() - 3;
	if (analysable_steps < 1)
		return;

	Float3 a(0.4);
	Float3 b(2.f);
	//LIMAENG::applyHyperpos(&a, &b);
	//testspace::testerfn(4);

	Float3* average_energy = new Float3[analysable_steps];
	
	for (int i = 0; i < simulation->getStep(); i++) {
		//printf("Step %d potE %f\n", i, simulation->potE_buffer[i]);
	}
	//exit(0);


																		// TODO: I think maybe i am missing 1 datapoint here? Something about only loading 99 steps in??
	hipMalloc(&traj_buffer_device, sizeof(Float3) * simulation->total_particles_upperbound * simulation->getStep());
	hipMalloc(&potE_buffer_device, sizeof(double) * simulation->total_particles_upperbound * simulation->getStep());
	hipMemcpy(traj_buffer_device, simulation->traj_buffer, sizeof(Float3) * simulation->total_particles_upperbound * simulation->getStep(), hipMemcpyHostToDevice);
	hipMemcpy(potE_buffer_device, simulation->potE_buffer, sizeof(double) * simulation->total_particles_upperbound * simulation->getStep(), hipMemcpyHostToDevice);


	Float3* average_solvent_energy = analyzeSolvateEnergy(simulation, analysable_steps);
	Float3* average_compound_energy = analyzeCompoundEnergy(simulation, analysable_steps);	//avg energy PER PARTICLE in compound

	hipFree(traj_buffer_device);
	hipFree(potE_buffer_device);


	
	for (int i = 0; i < analysable_steps; i++) {
		//printf("\n");
		//average_compound_energy[i].print('c');
		//average_solvent_energy[i].print('s');
		average_energy[i] = (average_solvent_energy[i] * simulation->box->n_solvents * 1 + average_compound_energy[i] * simulation->box->compounds[0].n_particles) * (1.f/ (simulation->box->n_solvents + simulation->box->compounds[0].n_particles));
		//average_energy[i].print('a');
	}
	

	printEnergies(average_energy, analysable_steps, simulation);


	delete [] average_solvent_energy, average_compound_energy, average_energy;


	printf("\n########## Finished analyzing energies ##########\n\n");

}

Float3* Analyzer::analyzeSolvateEnergy(Simulation* simulation, int n_steps) {
	dim3 block_dim(n_steps, 1, 1);
	Float3* average_solvent_energy = new Float3[n_steps];
	//Float3* host_data = new Float3[1 * n_steps];
	Float3* data_out;
	hipMalloc(&data_out, sizeof(Float3) * 1 * n_steps);


	//monitorSolventEnergyKernel <<< block_dim, simulation->n_solvents >>> (simulation->box, traj_buffer_device, potE_buffer_device, data_out);
	monitorSolventEnergyKernel << < block_dim, 256 >> > (simulation->box, traj_buffer_device, potE_buffer_device, data_out);
	hipDeviceSynchronize();
	//hipMemcpy(host_data, device_data, sizeof(Float3) * 1 * (n_steps), hipMemcpyDeviceToHost);
	hipMemcpy(average_solvent_energy, data_out, sizeof(Float3) * 1 * (n_steps), hipMemcpyDeviceToHost);

	for (int step = 0; step < n_steps; step++) {
		/*for (int i = 0; i < 256; i++) {
			if (host_data[i + step * 256].x > 10000)
				printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_solvent_energy[step] += host_data[i + step * 256];
		}*/
		average_solvent_energy[step] *= (1.f / simulation->n_solvents);
	}


	hipFree(data_out);
	//delete[] host_data;

	return average_solvent_energy;
}


Float3* Analyzer::analyzeCompoundEnergy(Simulation* simulation, int n_steps) {
	int n_datapoints = simulation->n_compounds * n_steps;
	printf("n steps: %d\n", n_steps);
	dim3 block_dim(n_steps, simulation->box->n_compounds, 1);
	Float3* average_compound_energy = new Float3[n_steps];
	Float3* host_data = new Float3[n_datapoints];
	Float3* data_out;
	hipMalloc(&data_out, sizeof(Float3) * n_datapoints);




	monitorCompoundEnergyKernel << < block_dim, MAX_COMPOUND_PARTICLES >> > (simulation->box, traj_buffer_device, potE_buffer_device, data_out);
	hipDeviceSynchronize();
	hipMemcpy(host_data, data_out, sizeof(Float3) * n_datapoints, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();


	printf("This energy analysis is only valid if just 1 compound exists!\n");
	for (int step = 0; step < n_steps; step++) {
		for (int i = 0; i < simulation->box->n_compounds; i++) {
			//if (host_data[i + step * 256].x > 10000)
				//printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_compound_energy[step] += host_data[i + step * simulation->box->n_compounds];
		}
		average_compound_energy[step] *= (1.f / (simulation->box->compounds[0].n_particles));
	}


	hipFree(data_out);
	delete[] host_data;

	return average_compound_energy;
}

void Analyzer::printEnergies(Float3* energy_data, int analysable_steps, Simulation* simulation) {
	string file_path_s = "D:\\Quantom\\energies_steps_" + to_string(analysable_steps) + ".bin";
	char* file_path;
	file_path = &file_path_s[0];
	cout << "Printing to file " << file_path << endl;

	FILE* file;
	fopen_s(&file, file_path, "wb");
	fwrite(energy_data, sizeof(Float3), analysable_steps, file);
	fclose(file);



	file_path_s = "D:\\Quantom\\temperatures_steps_" + to_string(analysable_steps) + ".bin";
	file_path = &file_path_s[0];
	cout << "Printing to file " << file_path << endl;

	fopen_s(&file, file_path, "wb");
	fwrite(simulation->temperature_buffer, sizeof(float), simulation->getStep()/STEPS_PER_THERMOSTAT, file);
	fclose(file);
}





/*

	std::ofstream myfile("D:\\Quantom\\energies_steps_" + to_string(analysable_steps) + ".csv");



	for (int i = 0; i < analysable_steps; i++) {
		for (int j = 0; j < 3; j++) {
			myfile << energy_data[i].at(j) << ';';
			//myfile << energy_data[j + i * 3] << ";";
		}
		myfile << "\n";
	}
	myfile.close();
*/


















