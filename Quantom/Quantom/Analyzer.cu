#include "hip/hip_runtime.h"
#include "Analyzer.cuh"







void __device__ applyHyperposA(Float3* static_particle, Float3* movable_particle) {
	for (int i = 0; i < 3; i++) {
		*movable_particle->placeAt(i) += BOX_LEN * ((static_particle->at(i) - movable_particle->at(i)) > BOX_LEN_HALF);
		*movable_particle->placeAt(i) -= BOX_LEN * ((static_particle->at(i) - movable_particle->at(i)) < -BOX_LEN_HALF);	// use at not X!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	}
}

void __global__ monitorCompoundEnergyKernel(Box* box, Float3* data_out) {		// everything here breaks if not all compounds are identical in particle count and particle mass!!!!!!!
	__shared__ Float3 energy[PARTICLES_PER_COMPOUND];
	__shared__ Compound compound;
	//__shared__ float mass;	// uhhh, this is not nedcessary the same for all particles in compound.


	const int step = blockIdx.x + 1;
	const int compound_index = blockIdx.y;
	energy[threadIdx.x] = Float3(0, 0, 0);

	if (threadIdx.x == 0) {
		//printf("index: %d\n", compound_index + (step - 1) * N_MONITORBLOCKS_PER_STEP);
		data_out[compound_index + (step - 1) * box->n_compounds] = Float3(0, 0, 0);
		//mass = box->compounds[compound_index].particles[0]
		compound = box->compounds[compound_index];
	}

	__syncthreads();

	if (threadIdx.x >= compound.n_particles) {
		return;
	}
	__syncthreads();


	double potE = box->potE_buffer[threadIdx.x + compound_index * PARTICLES_PER_COMPOUND + step * box->total_particles];	

	Float3 pos_tsub1 = box->trajectory[threadIdx.x + compound_index * PARTICLES_PER_COMPOUND + (step - 1) * box->total_particles];
	Float3 pos_tadd1 = box->trajectory[threadIdx.x + compound_index * PARTICLES_PER_COMPOUND + (step + 1) * box->total_particles];
	applyHyperposA(&pos_tadd1, &pos_tsub1);
	

	double vel = (pos_tadd1 - pos_tsub1).len() * 0.5f / box->dt;
	double kinE = 0.5 * compound.particles[threadIdx.x].mass * vel * vel;

	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();
	for (int i = 1; i < PARTICLES_PER_COMPOUND; i *= 2) {	// Distributed averaging
		if ((threadIdx.x + i) < PARTICLES_PER_COMPOUND) {
			energy[threadIdx.x] = (energy[threadIdx.x] + energy[threadIdx.x + i]);// *0.5f;	// easier to just divide by sum of solvents at host
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		data_out[compound_index + (step - 1) * box->n_compounds] = energy[0];
	}
}





void __global__ monitorSolventEnergyKernel(Box* box, Float3* data_out) {
	__shared__ Float3 energy[THREADS_PER_MONITORBLOCK];
	int solvent_index = threadIdx.x + blockIdx.y * THREADS_PER_MONITORBLOCK;
	int step = blockIdx.x + 1;
	int compounds_offset = box->n_compounds * PARTICLES_PER_COMPOUND;


	energy[threadIdx.x] = Float3(0, 0, 0);
	if (solvent_index >= box->n_solvents) {
		return;
	}


	if (threadIdx.x == 0) {
		data_out[blockIdx.y + (step - 1) * N_MONITORBLOCKS_PER_STEP] = Float3(0, 0, 0);
	}

	Float3 pos_tsub1 = box->trajectory[compounds_offset + solvent_index + (step - 1) * box->total_particles];
	Float3 pos_tadd1 = box->trajectory[compounds_offset + solvent_index + (step + 1) * box->total_particles];
	applyHyperposA(&pos_tadd1, &pos_tsub1);


	double potE = box->potE_buffer[compounds_offset + solvent_index + step * box->total_particles];

	
	double vel = (pos_tadd1 - pos_tsub1).len() * 0.5f / box->dt;
	if (vel > 10'000) {
		printf("step %04d solvate %04dvel:  %f\n", step, solvent_index, vel);
		//pos_tadd1.print('a');
		//printf("analyzer index %d\n", compounds_offset + solvent_index + (step - 1) * box->total_particles);
		//pos_tsub1.print('s');
	}
	if (potE > 200'000) {
		printf("step %04d solvate %04d pot %f\n", step, solvent_index, potE);
	}
	
		
	double kinE = 0.5 * SOLVENT_MASS * vel * vel;


	double totalE = potE + kinE;

	energy[threadIdx.x] = Float3(potE, kinE, totalE);
	__syncthreads();

	for (int i = 1; i < THREADS_PER_MONITORBLOCK; i *= 2) {	// Distributed averaging
		if ((threadIdx.x + i) < THREADS_PER_MONITORBLOCK) {
			energy[threadIdx.x] = (energy[threadIdx.x] + energy[threadIdx.x + i]);// *0.5f;	// easier to just divide by sum of solvents at host
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		//energy[0].print('b');
		data_out[blockIdx.y + (step-1) * N_MONITORBLOCKS_PER_STEP] = energy[0];
		//data_out[blockIdx.y + (step - 1) * 256] = Float3(1,2,3);
		//data_out[blockIdx.y + step * 256].print('g');
	}
	
}



void Analyzer::analyzeEnergy(Simulation* simulation) {	// Calculates the avg J/mol // calculate energies separately for compounds and solvents. weigh averages based on amount of each
	int analysable_steps = simulation->n_steps - 2;
	if (analysable_steps < 1)
		return;

	Float3* average_energy = new Float3[analysable_steps];
	

	Float3* average_solvent_energy = analyzeSolvateEnergy(simulation, analysable_steps);
	Float3* average_compound_energy = analyzeCompoundEnergy(simulation, analysable_steps);	//avg energy PER PARTICLE in compound

	for (int i = 0; i < analysable_steps; i++) {
		average_energy[i] = (average_solvent_energy[i] * simulation->box->n_solvents * 1 + average_compound_energy[i] * simulation->box->n_compounds * PARTICLES_PER_COMPOUND) * (1.f/ simulation->box->total_particles);
	}


	printEnergies(average_energy, analysable_steps);


	delete [] average_solvent_energy, average_compound_energy, average_energy;


	printf("\n########## Finished analyzing energies ##########\n\n");

}

Float3* Analyzer::analyzeSolvateEnergy(Simulation* simulation, int n_steps)
{
	// Solvent energies first //
	dim3 block_dim(n_steps, 256, 1);
	Float3* average_solvent_energy = new Float3[n_steps];
	Float3* host_data = new Float3[256 * n_steps];
	Float3* device_data;
	hipMalloc(&device_data, sizeof(Float3) * 256 * n_steps);


	monitorSolventEnergyKernel <<< block_dim, 256 >>> (simulation->box, device_data);
	hipDeviceSynchronize();
	hipMemcpy(host_data, device_data, sizeof(Float3) * 256 * (n_steps), hipMemcpyDeviceToHost);

	for (int step = 0; step < n_steps; step++) {
		for (int i = 0; i < 256; i++) {
			if (host_data[i + step * 256].x > 10000)
				printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_solvent_energy[step] += host_data[i + step * 256];
		}
		average_solvent_energy[step] *= (1.f / simulation->box->n_solvents);
	}


	hipFree(device_data);
	delete[] host_data;

	return average_solvent_energy;
}

Float3* Analyzer::analyzeCompoundEnergy(Simulation* simulation, int n_steps)
{
	int n_datapoints = simulation->box->n_compounds * n_steps;
	printf("n steps: %d\n", n_steps);
	dim3 block_dim(n_steps, simulation->box->n_compounds, 1);
	Float3* average_compound_energy = new Float3[n_steps];
	Float3* host_data = new Float3[n_datapoints];
	Float3* device_data;
	hipMalloc(&device_data, sizeof(Float3) * n_datapoints);

	monitorCompoundEnergyKernel << < block_dim, PARTICLES_PER_COMPOUND >> > (simulation->box, device_data);
	hipDeviceSynchronize();
	hipMemcpy(host_data, device_data, sizeof(Float3) * n_datapoints, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	int nnn = simulation->box->n_compounds;
	printf("What %d\n", simulation->box->n_compounds);
	for (int step = 0; step < n_steps; step++) {
		for (int i = 0; i < simulation->box->n_compounds; i++) {
			//if (host_data[i + step * 256].x > 10000)
				//printf("Block: %d energy: %f\n", i, host_data[i + step * 256].x);
			average_compound_energy[step] += host_data[i + step * simulation->box->n_compounds];
		}
		average_compound_energy[step] *= (1.f / (simulation->box->n_compounds * PARTICLES_PER_COMPOUND));
	}


	hipFree(device_data);
	delete[] host_data;

	return average_compound_energy;
}


void Analyzer::printEnergies(Float3* energy_data, int analysable_steps) {
	std::ofstream myfile("D:\\Quantom\\energies.csv");


	for (int i = 0; i < analysable_steps; i++) {
		for (int j = 0; j < 3; j++) {
			myfile << energy_data[i].at(j) << ';';
			//myfile << energy_data[j + i * 3] << ";";
		}
		myfile << "\n";
	}
	myfile.close();
}
























